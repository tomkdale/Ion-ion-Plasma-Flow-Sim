#include "hip/hip_runtime.h"
//-----------------------------------------------------
//Engine code analysing ion-ion flow
//Written by Dr. Kamesh Sankaran, Thomas Dale and Travis Widmer
//Summer 2017
//-----------------------------------------------------

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include "PreProccessors.h"
#include "tMins.h"
#include "VoltagesGrounded.h"
#include "VoltagesZeroGradient.h"
#include "Conservatives.h"
#include "Temperatures.h"
#include "Fluxes.h"
#include "eFields.h"
#include <random>


int main() {
	//-----------------------------------------
	clock_t start = clock(); // real run time
	double t = 0.0; //current Engine time
	//-----------------------------------------
	//Do necessary calculations for code setup
	int numCenters = numSegR *numSegZ;
	int numEdges = (numSegR + 1)*(numSegZ + 1);
	int numRFluxes = (numSegR + 1) * numSegZ;
	int numZFluxes = numSegR * (numSegZ + 1);
	double dR = length_R / numSegR;
	double dZ = length_Z / numSegZ;
	size_t sizeCenters = numCenters * sizeof(double);
	size_t sizeEdges = numEdges * sizeof(double);
	size_t sizeBoolEdges = numEdges * sizeof(bool);
	size_t sizeFluxR = numRFluxes * sizeof(double);
	size_t sizeFluxZ = numZFluxes * sizeof(double);
	dim3 blocksPerGridCenters((numSegZ / threadsPerBlock2D) + 1, (numSegR / threadsPerBlock2D) + 1);
	dim3 blocksPerGridEdges(((numSegZ +1) / threadsPerBlock2D) + 1, ((numSegR+1)/ threadsPerBlock2D) + 1);
	dim3 blocksPerGridFluxRs((numSegZ / threadsPerBlock2D) + 1, ((numFluxR) / threadsPerBlock2D) + 1);
	dim3 blocksPerGridFluxZs(((numFluxZ) / threadsPerBlock2D) + 1, (numSegR / threadsPerBlock2D) + 1);
	dim3 threadsPerBlockDim3(threadsPerBlock2D, threadsPerBlock2D);
	dim3 blocksPerGridCompressed((numSegZ / threadsPerBlock2D) + 1, ((numSegR / 2) / threadsPerBlock2D) + 1);
	//-----------------------------------------

	//-----------------------------------------
	//create array variables and allocate memory
	double *tempP, *tempN, *volt, *densityP, *densityN, *velocityRP, *velocityRN, *velocityZP, *velocityZN, *tMins, *eFieldR, *eFieldZ, *spectralRadiusR, *spectralRadiusZ, *collisionFreq;
	bool *redBlackConvergence;
	tempP = (double*)malloc(sizeCenters);
	tempN = (double*)malloc(sizeCenters);
	volt = (double*)malloc(sizeEdges);
	densityN = (double*)malloc(sizeCenters);
	densityP = (double*)malloc(sizeCenters);
	velocityRP = (double*)malloc(sizeCenters);
	velocityRN = (double*)malloc(sizeCenters);
	velocityZP = (double*)malloc(sizeCenters);
	velocityZN = (double*)malloc(sizeCenters);
	tMins = (double*)malloc(sizeCenters);
	eFieldR = (double*)malloc(sizeCenters);
	eFieldZ = (double*)malloc(sizeCenters);
	spectralRadiusR = (double*)malloc(sizeCenters);
	spectralRadiusZ = (double*)malloc(sizeCenters);
	collisionFreq = (double*)malloc(sizeCenters);
	redBlackConvergence = (bool*)malloc(sizeBoolEdges);


	if (fresh == 1) {// Create fresh start initial variables
		for (int it = 0; it < numEdges; it++)
			volt[it] = 0.;
		for (int it = 0; it < numCenters; it++)
			tempP[it] = thrusterTemp;
		for (int it = 0; it < numCenters; it++)
			tempN[it] = thrusterTemp;
		for (int it = 0; it < numCenters; it++)
			densityP[it] = minDensity;
		for (int it = 0; it < numCenters; it++)
			densityN[it] = minDensity;
		for (int it = 0; it < numCenters; it++)//velocity of positive ions in r direction
			velocityRP[it] = 0;
		for (int it = 0; it < numCenters; it++)// velocity of negative in r direction
			velocityRN[it] = 0;
		for (int it = 0; it < numCenters; it++)// velocity of positive ions in z direction
			velocityZP[it] = 0;
		for (int it = 0; it < numCenters; it++)// velocity of negative ions in z direction
			velocityZN[it] = 0;
	}
	else {//assign initial variables from initialValues.txt
		FILE* arrays;
		arrays = fopen("initialValues.txt", "r");
		fscanf(arrays, "%*s");
		for (int it = 0; it < numEdges; it++)
			fscanf(arrays, "%le", &volt[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &tempP[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &tempN[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &densityP[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &densityN[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &velocityRP[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &velocityRN[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &velocityZP[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &velocityZN[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &eFieldR[it]);
		fscanf(arrays, "%*s");
		for (int it = 0; it < numCenters; it++)
			fscanf(arrays, "%le", &eFieldZ[it]);
		fscanf(arrays, "%*s");
		fscanf(arrays, "%le", &t);
		fclose(arrays);
	}

	//initialize collision frequency
	for (int it = 0; it < numCenters; it++)
	{
		//collisionFreq[it] = 1e5; //TODO solve problem using nonzero value for collisionfreq once test case is verified
		collisionFreq[it] = 0.;
	}
	//--------------------------------------------
	

	//-----------------------------------------
	//intialize Device side variables
	double *d_oldTempP, *d_oldTempN, *d_oldVolt, *d_oldDensityP, *d_oldDensityN, *d_oldVelocityRP, *d_oldVelocityRN, *d_oldVelocityZP, *d_oldVelocityZN, *d_tMins, *d_eFieldR, *d_eFieldZ,*d_spectralRadiusR, *d_spectralRadiusZ, *d_collisionFreq, *d_newTempP, *d_newTempN, *d_newVolt, *d_newDensityP, *d_newDensityN, *d_newVelocityRP, *d_newVelocityRN, *d_newVelocityZP, *d_newVelocityZN, *fluxR1,*fluxR2,*fluxR3,*fluxR4,*fluxR5,*fluxR6,*fluxZ1,*fluxZ2,*fluxZ3,*fluxZ4,*fluxZ5,*fluxZ6;
	bool *d_redBlackConvergence;

	hipMalloc(&d_tMins, sizeCenters);
	hipMalloc(&d_oldTempP, sizeCenters);
	hipMalloc(&d_oldTempN, sizeCenters);
	hipMalloc(&d_oldVolt, sizeEdges);
	hipMalloc(&d_oldDensityP, sizeCenters);
	hipMalloc(&d_oldDensityN, sizeCenters);
	hipMalloc(&d_oldVelocityRP, sizeCenters);
	hipMalloc(&d_oldVelocityRN, sizeCenters);
	hipMalloc(&d_oldVelocityZP, sizeCenters);
	hipMalloc(&d_oldVelocityZN, sizeCenters);
	hipMalloc(&d_eFieldR, sizeCenters);
	hipMalloc(&d_eFieldZ, sizeCenters);
	hipMalloc(&d_spectralRadiusR, sizeCenters);
	hipMalloc(&d_spectralRadiusZ, sizeCenters);
	hipMalloc(&d_collisionFreq, sizeCenters);

	hipMalloc(&d_newTempP, sizeCenters);
	hipMalloc(&d_newTempN, sizeCenters);
	hipMalloc(&d_newVolt , sizeEdges);
	hipMalloc(&d_newDensityP, sizeCenters);
	hipMalloc(&d_newDensityN, sizeCenters);
	hipMalloc(&d_newVelocityRP, sizeCenters);
	hipMalloc(&d_newVelocityRN, sizeCenters);
	hipMalloc(&d_newVelocityZP, sizeCenters);
	hipMalloc(&d_newVelocityZN, sizeCenters);
	hipMalloc(&d_redBlackConvergence,sizeBoolEdges);

	hipMalloc(&fluxR1, sizeFluxR);
	hipMalloc(&fluxR2, sizeFluxR);
	hipMalloc(&fluxR3, sizeFluxR);
	hipMalloc(&fluxR4, sizeFluxR);
	hipMalloc(&fluxR5, sizeFluxR);
	hipMalloc(&fluxR6, sizeFluxR);

	hipMalloc(&fluxZ1, sizeFluxZ);
	hipMalloc(&fluxZ2, sizeFluxZ);
	hipMalloc(&fluxZ3, sizeFluxZ);
	hipMalloc(&fluxZ4, sizeFluxZ);
	hipMalloc(&fluxZ5, sizeFluxZ);
	hipMalloc(&fluxZ6, sizeFluxZ);


	hipMemcpy(d_oldVolt, volt, sizeEdges, hipMemcpyHostToDevice);
	hipMemcpy(d_oldTempP, tempP, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldTempN, tempN, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldDensityP, densityP, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldDensityN, densityN, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldVelocityRP, velocityRP, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldVelocityRN, velocityRN, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldVelocityZP, velocityZP, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_oldVelocityZN,velocityZN, sizeCenters, hipMemcpyHostToDevice);
	hipMemcpy(d_collisionFreq, collisionFreq, sizeCenters, hipMemcpyHostToDevice);

	//if starting from initialValues.txt results copy data to device
	if (fresh==0)
	{
		hipMemcpy(d_eFieldR, eFieldR, sizeCenters, hipMemcpyHostToDevice);
		hipMemcpy(d_eFieldZ, eFieldZ, sizeCenters, hipMemcpyHostToDevice);
		printf("Starting from previous results.\nCopied electric fields to the GPU.\n");
	}
	//-----------------------------------------


	//-----------------------------------------
	//Start Actual Calculation portion
	unsigned int counter = 0;//time loop counter
	bool converged = false;
	if (fresh == 1) {
		do {//first Red-Black method loop to solve for initial voltages
			calc_voltagesRedZeroGradient << < blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_oldDensityP, d_oldDensityN, dR, dZ, d_redBlackConvergence);
			calc_voltagesBlackZeroGradient << <blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_oldDensityP, d_oldDensityN, dR, dZ, d_redBlackConvergence);
			hipMemcpy(redBlackConvergence, d_redBlackConvergence, sizeBoolEdges, hipMemcpyDeviceToHost);
			hipMemcpy(d_oldVolt, d_newVolt, sizeEdges, hipMemcpyDeviceToDevice);
			for (int it = 0; it < numEdges; it++) {
				if (!redBlackConvergence[it]) break;
				if (it == numEdges - 1) converged = true;
			}
			counter++;
		} while (!converged);
		printf("First calc_voltage call took %d steps.\n", counter);
	}

	else
	{
		hipMemcpy(d_newVolt, d_oldVolt, sizeEdges, hipMemcpyDeviceToDevice);
	}


	counter = 0; //Reset the counter

	while (t < Total_Time) { //UNCOMMENT TO RUN TO TOTAL TIME
	//while (counter < 10000) {  //UNCOMMENT TO RUN BY COUNTER. NOTE: REMEMBER TO SET TOTAL TIME much higher than count would every reach to avoid crash
		//Calculate the electric field from voltage
		calc_eFields <<<blocksPerGridCenters, threadsPerBlockDim3 >>> (d_eFieldR, d_eFieldZ, d_newVolt, dR, dZ);

		//Find the smallest time step needed at any location in the domain
		calc_tmins <<<blocksPerGridCenters, threadsPerBlockDim3 >>> (d_oldTempP, d_oldTempN, d_oldVelocityRP, d_oldVelocityRN, d_oldVelocityZP, d_oldVelocityZN, d_tMins, d_spectralRadiusR, d_spectralRadiusZ, dR, dZ);
		hipMemcpy(tMins, d_tMins, sizeCenters, hipMemcpyDeviceToHost);
		for (int it = 0; it < numCenters; it++) {//move smallest tMins to tMins[0]
			if (tMins[0] > tMins[it]) tMins[0] = tMins[it];
		}
		double timeStep = tMins[0];

		//if on last step, shorten timestep to end exactly on total_time
		if (timeStep + t > Total_Time) {
			timeStep = Total_Time - t;
		}
		//calc fluxes
		calc_fluxesR << <blocksPerGridFluxRs, threadsPerBlockDim3 >> > (d_oldTempP, d_oldTempN, d_oldDensityP, d_oldDensityN, d_oldVelocityRP, d_oldVelocityRN, d_oldVelocityZP, d_oldVelocityZN, d_eFieldR, d_eFieldZ, fluxR1, fluxR2, fluxR3, fluxR4, fluxR5, fluxR6, d_spectralRadiusR, dR, dZ);
		calc_fluxesZ << <blocksPerGridFluxZs, threadsPerBlockDim3 >> > (d_oldTempP, d_oldTempN, d_oldDensityP, d_oldDensityN, d_oldVelocityRP, d_oldVelocityRN, d_oldVelocityZP, d_oldVelocityZN, d_eFieldR, d_eFieldZ, fluxZ1, fluxZ2, fluxZ3, fluxZ4, fluxZ5, fluxZ6, d_spectralRadiusZ, dR, dZ, t);


		//Calculate temperatures ... for now, this is redundant since they are held constant
		calc_temperatures <<<blocksPerGridCenters, threadsPerBlockDim3 >>> (d_newTempP, d_oldTempP, d_newTempN, d_oldTempN);

		//Calculate the conservation of mass and momentum
		calc_conservatives<<<blocksPerGridCenters, threadsPerBlockDim3 >>> (fluxR1, fluxR2, fluxR3, fluxR4, fluxR5, fluxR6, fluxZ1, fluxZ2, fluxZ3, fluxZ4, fluxZ5, fluxZ6, d_newTempP, d_newTempN, d_newDensityP, d_oldDensityP, d_newDensityN, d_oldDensityN, d_newVelocityRP, d_oldVelocityRP, d_newVelocityRN, d_oldVelocityRN, d_newVelocityZP, d_oldVelocityZP, d_newVelocityZN, d_oldVelocityZN, d_eFieldR, d_eFieldZ, d_collisionFreq, dR, dZ, timeStep);

		//Use the Red-Black iterative method to solve for voltages based on the new charge densities
		converged = false;
		do {
			if (grounded == 0) {
				calc_voltagesRedZeroGradient << < blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_newDensityP, d_newDensityN, dR, dZ, d_redBlackConvergence);
				calc_voltagesBlackZeroGradient << <blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_newDensityP, d_newDensityN, dR, dZ, d_redBlackConvergence);
			}
			else {
				calc_voltagesRedGrounded << < blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_newDensityP, d_newDensityN, dR, dZ, d_redBlackConvergence);
				calc_voltagesBlackGrounded << <blocksPerGridCompressed, threadsPerBlockDim3 >> > (d_newVolt, d_oldVolt, d_newDensityP, d_newDensityN, dR, dZ, d_redBlackConvergence);
			}
			hipMemcpy(redBlackConvergence, d_redBlackConvergence, sizeBoolEdges, hipMemcpyDeviceToHost);
			hipMemcpy(d_oldVolt, d_newVolt, sizeEdges, hipMemcpyDeviceToDevice);
			for (int it = 0; it < numEdges; it++) {
				if (!redBlackConvergence[it]) break;
				if (it == numEdges - 1) converged = true;
			}
		} while (!converged);

		//update olds from new
		hipMemcpy(d_oldTempP, d_newTempP, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldTempN, d_newTempN, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldDensityP, d_newDensityP, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldDensityN, d_newDensityN, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldVelocityRP, d_newVelocityRP, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldVelocityRN, d_newVelocityRN, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldVelocityZP, d_newVelocityZP, sizeCenters, hipMemcpyDeviceToDevice);
		hipMemcpy(d_oldVelocityZN, d_newVelocityZN, sizeCenters, hipMemcpyDeviceToDevice);

		//periodically (e.g., every 1000 steps) display a screen message
		counter++;
		t += timeStep;
		if (counter % 10 == 0) {
			printf("Excecuted %d steps using timeStep=%le to get to t=%le.\n", counter, timeStep, t);
		}
	}//end of calculation while loop

	printf("Timestep loop ran for %d steps.\n", counter);
	//-----------------------------------------
	
	//calculate eFields one more time so they update to the most recent voltage
	calc_eFields <<<blocksPerGridCenters, threadsPerBlockDim3 >>> (d_eFieldR, d_eFieldZ, d_newVolt, dR, dZ);
	//-----------------------------------------
	//copy final GPU data back to CPU
	hipMemcpy(volt,d_newVolt, sizeEdges, hipMemcpyDeviceToHost);
	hipMemcpy(tempP,d_newTempP, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(tempN,d_newTempN, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(densityP,d_newDensityP, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(densityN,d_newDensityN, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(velocityRP,d_newVelocityRP, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(velocityRN,d_newVelocityRN, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(velocityZP,d_newVelocityZP, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(velocityZN,d_newVelocityZN, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(eFieldR, d_eFieldR, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(eFieldZ, d_eFieldZ, sizeCenters, hipMemcpyDeviceToHost);

	hipMemcpy(spectralRadiusR, d_spectralRadiusR, sizeCenters, hipMemcpyDeviceToHost);
	hipMemcpy(spectralRadiusZ, d_spectralRadiusZ, sizeCenters, hipMemcpyDeviceToHost);

	//-----------------------------------------
	
	//-----------------------------------------
	//print results to results.txt
	FILE* finals;
	finals = fopen("results.txt", "w");
	fprintf(finals,"Voltage ");
	for (int it = 0; it < numEdges; it++)
		fprintf(finals, "%le ",volt[it]);
	fprintf(finals, "\nTemperature_Positive ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", tempP[it]);
	fprintf(finals, "\nTemperature_Negative ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", tempN[it]);
	fprintf(finals, "\nDensity_Positive ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", densityP[it]);
	fprintf(finals, "\nDensity_Negative ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", densityN[it]);
	fprintf(finals, "\nVelocity_Positive_Radial ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", velocityRP[it]);
	fprintf(finals, "\nVelocity_Negative_Radial ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", velocityRN[it]);
	fprintf(finals, "\nVelocity_Positive_ZDirection ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", velocityZP[it]);
	fprintf(finals, "\nVelocity_Negative_ZDirection ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", velocityZN[it]);
	fprintf(finals, "\nEField_Radial ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", eFieldR[it]);
	fprintf(finals, "\nEfield_Zdirection ");
	for (int it = 0; it < numCenters; it++)
		fprintf(finals, "%le ", eFieldZ[it]);
	fprintf(finals, "\nTime_Reached: %le", t);
	fclose(finals);
	clock_t end = clock();
	printf("Program took %le to execute.\n", (float)(end - start) /CLOCKS_PER_SEC);
	//-----------------------------------------

	//As your mom taught you, clean up after yourself
	hipFree(d_oldTempP);
	hipFree(d_oldTempN);
	hipFree(d_oldVolt);
	hipFree(d_oldDensityP);
	hipFree(d_oldDensityN);
	hipFree(d_oldVelocityRP);
	hipFree(d_oldVelocityRN);
	hipFree(d_oldVelocityZP);
	hipFree(d_oldVelocityZN);
	hipFree(d_tMins);
	hipFree(d_eFieldR);
	hipFree(d_eFieldZ);
	hipFree(d_spectralRadiusR);
	hipFree(d_spectralRadiusZ);
	hipFree(d_collisionFreq);
	hipFree(d_newTempP);
	hipFree(d_newTempN);
	hipFree(d_newVolt);
	hipFree(d_newDensityP);
	hipFree(d_newDensityN);
	hipFree(d_newVelocityRP);
	hipFree(d_newVelocityRN);
	hipFree(d_newVelocityZP);
	hipFree(d_newVelocityZN);
	hipFree(d_eFieldR);
	hipFree(d_eFieldZ);
	hipFree(d_redBlackConvergence);
	hipFree(fluxR1);
	hipFree(fluxR2);
	hipFree(fluxR3);
	hipFree(fluxR4);
	hipFree(fluxR5); 
	hipFree(fluxR6);
	hipFree(fluxZ1);
	hipFree(fluxZ2);
	hipFree(fluxZ3);
	hipFree(fluxZ4);
	hipFree(fluxZ5);
	hipFree(fluxZ6);

	return 0;
}//DONE
